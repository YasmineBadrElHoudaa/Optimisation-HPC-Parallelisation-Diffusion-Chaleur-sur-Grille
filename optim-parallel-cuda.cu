#include "hip/hip_runtime.h"
%%cuda
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

#define TAILLE 100
#define ITERATIONS 100
#define N 16

__global__ void calculer_nouvelle_grille(double* grille, double* nouvelle_grille, int taille) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i > 0 && i < taille - 1 && j > 0 && j < taille - 1) {
        nouvelle_grille[i * taille + j] = 0.25 * (grille[(i - 1) * taille + j] + grille[(i + 1) * taille + j] +
                                                 grille[i * taille + (j - 1)] + grille[i * taille + (j + 1)]) +
                                         10 * sin(i * 0.01) * cos(j * 0.01);
    }
}

__global__ void mettre_a_jour_grille(double* grille, double* nouvelle_grille, int taille) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i > 0 && i < taille - 1 && j > 0 && j < taille - 1) {
        grille[i * taille + j] = nouvelle_grille[i * taille + j];
    }
}

int main() {
    double* h_grille = (double*)malloc(TAILLE * TAILLE * sizeof(double));
    double* h_nouvelle_grille = (double*)malloc(TAILLE * TAILLE * sizeof(double));

    srand(time(NULL));
    for (int i = 0; i < TAILLE; i++) {
        for (int j = 0; j < TAILLE; j++) {
            h_grille[i * TAILLE + j] = rand() % 100;
        }
    }

    double *d_grille, *d_nouvelle_grille;
    hipMalloc((void**)&d_grille, TAILLE * TAILLE * sizeof(double));
    hipMalloc((void**)&d_nouvelle_grille, TAILLE * TAILLE * sizeof(double));

    hipMemcpy(d_grille, h_grille, TAILLE * TAILLE * sizeof(double), hipMemcpyHostToDevice);
      // N*1 = nombre de threads par Bloc
    dim3 threadsParBloc(N, 1);
   
    dim3 blocsParGrille((TAILLE + threadsParBloc.x - 1) / threadsParBloc.x,
                        (TAILLE + threadsParBloc.y - 1) / threadsParBloc.y);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    for (int t = 0; t < ITERATIONS; t++) {
        calculer_nouvelle_grille<<<blocsParGrille, threadsParBloc>>>(d_grille, d_nouvelle_grille, TAILLE);
        hipDeviceSynchronize();

        mettre_a_jour_grille<<<blocsParGrille, threadsParBloc>>>(d_grille, d_nouvelle_grille, TAILLE);
        hipDeviceSynchronize();

        if (t % 100 == 0) {
            printf("Iteration %d/%d terminee\n", t, ITERATIONS);
        }
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Temps d'execution: %.2f secondes\n", milliseconds / 1000.0);

    hipMemcpy(h_grille, d_grille, TAILLE * TAILLE * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_grille);
    hipFree(d_nouvelle_grille);

    free(h_grille);
    free(h_nouvelle_grille);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
